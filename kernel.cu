#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
//CUDA RunTime API
#include <hip/hip_runtime.h>
using namespace std;
#define THREAD_NUM 1024
#define MATRIX_SIZE 2000

//�����߳̿�ĸ���������ȡ����
const int blocks_num = (MATRIX_SIZE * MATRIX_SIZE + THREAD_NUM - 1) / THREAD_NUM;

//CUDA ��ʼ��
bool InitCUDA()
{
    int count;

    //ȡ��֧��Cuda��װ�õ���Ŀ
    hipGetDeviceCount(&count);
    if (count == 0)
    {
        fprintf(stderr, "There is no device.\n");

        return false;
    }
    int i;
    for (i = 0; i < count; i++)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        if (hipGetDeviceProperties(&prop, i) == hipSuccess)
        {
            if (prop.major >= 1)
            {
                break;
            }
        }
    }
    if (i == count)
    {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }
    hipSetDevice(i);
    return true;
}

//�����ʼ��
void matgen(int* a, int n)
{
    int i, j;
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            a[i * n + j] = i + j;
        }
    }
}

// __global__ ���� ���м������˷�
__global__ static void matMultCUDA(const int* a, const int* b, int* c, int n)
{
    //��ʾĿǰ�� thread �ǵڼ��� thread���� 0 ��ʼ���㣩
    const int tid = threadIdx.x;

    //��ʾĿǰ�� thread ���ڵڼ��� block���� 0 ��ʼ���㣩
    const int bid = blockIdx.x;

    //�� bid �� tid �������� thread Ӧ�ü���� row �� column
    const int idx = bid * THREAD_NUM + tid;
    const int row = idx / n;
    const int column = idx % n;
    int i;
    if (row < n && column < n)
    {
        int t = 0;
        for (i = 0; i < n; i++)
        {
            t += a[row * n + i] * b[i * n + column];
        }
        c[row * n + column] = t;
    }
}

// ������
int main()
{
    //CUDA ��ʼ��
    if (!InitCUDA()) return 0;

    //�������
    int* a, * b, * c;
    int n = MATRIX_SIZE;

    //�����ڴ�
    a = (int*)malloc(sizeof(int) * n * n);
    b = (int*)malloc(sizeof(int) * n * n);
    c = (int*)malloc(sizeof(int) * n * n);

    //���ɾ���
    matgen(a, n);
    matgen(b, n);

    /*�����ݸ��Ƶ��Կ��ڴ���*/
    int* cuda_a, * cuda_b, * cuda_c;

    //hipMalloc ȡ��һ���Կ��ڴ� 
    hipMalloc((void**)&cuda_a, sizeof(int) * n * n);
    hipMalloc((void**)&cuda_b, sizeof(int) * n * n);
    hipMalloc((void**)&cuda_c, sizeof(int) * n * n);

    //hipMemcpy �������ľ����Ƶ��Կ��ڴ���
    //hipMemcpyHostToDevice - ���ڴ渴�Ƶ��Կ��ڴ�
    //hipMemcpyDeviceToHost - ���Կ��ڴ渴�Ƶ��ڴ�
    hipMemcpy(cuda_a, a, sizeof(int) * n * n, hipMemcpyHostToDevice);
    hipMemcpy(cuda_b, b, sizeof(int) * n * n, hipMemcpyHostToDevice);

    // ��ʱ
    hipEvent_t gpuStart, gpuFinish;
    float elapsedTime;
    hipEventCreate(&gpuStart);
    hipEventCreate(&gpuFinish);
    hipEventRecord(gpuStart, 0);

    // ��CUDA ��ִ�к��� �﷨����������<<<block ��Ŀ, thread ��Ŀ, shared memory ��С>>>(����...);
    matMultCUDA << < blocks_num, THREAD_NUM, 0 >> > (cuda_a, cuda_b, cuda_c, n);
    hipEventRecord(gpuFinish, 0);
    hipEventSynchronize(gpuStart);
    hipEventSynchronize(gpuFinish);
    hipEventElapsedTime(&elapsedTime, gpuStart, gpuFinish);
    printf("\nThe runing time of GPU on Mat Multiply is %f seconds.\n", elapsedTime / 1000.0);

    //hipMemcpy ��������Դ��и��ƻ��ڴ�
    hipMemcpy(c, cuda_c, sizeof(int) * n * n, hipMemcpyDeviceToHost);

    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);
    return 0;
}